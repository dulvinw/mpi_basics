
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel to double an integer value
__global__ void doubleValue(int *value) {
    *value *= 2;
}

int main() {
    int h_value = 10; // host value
    int *d_value; // device value pointer

    // Allocate device memory
    hipMalloc((void**)&d_value, sizeof(int));

    // Copy host value to device value
    hipMemcpy(d_value, &h_value, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to double the value
    doubleValue<<<1, 1>>>(d_value);

    // Copy result back to host
    hipMemcpy(&h_value, d_value, sizeof(int), hipMemcpyDeviceToHost);

    // Print the doubled value
    printf("Doubled value is: %d\n", h_value);

    // Free device memory
    hipFree(d_value);

    return 0;
}

