
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel to double the values of an array
__global__ void doubleArray(int *array, int arraySize) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < arraySize) {
        array[idx] = __mul24(array[idx], 2);
    }
}

int main() {
    const int arraySize = 10;
    int h_array[arraySize]; // host array
    int *d_array; // device array pointer

    // Initialize host array with values 1 to 10
    for (int i = 0; i < arraySize; ++i) {
        h_array[i] = i + 1;
    }

    // Allocate device memory
    hipMalloc((void**)&d_array, arraySize * sizeof(int));

    // Copy host array to device array
    hipMemcpy(d_array, h_array, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to double the array elements
    // Using 1 block with arraySize threads for simplicity
    doubleArray<<<1, arraySize>>>(d_array, arraySize);

    // Copy result back to host
    hipMemcpy(h_array, d_array, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // Print the doubled values
    printf("Doubled array values are: ");
    for (int i = 0; i < arraySize; ++i) {
        printf("%d ", h_array[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_array);

    return 0;
}

